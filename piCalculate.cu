// nvcc -ccbin "D:\Program Files (x86)\Microsoft Visual Studio 11.0\VC\bin" piCalculate.cu -o piCalculate.exe

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>

#define MAX_CUDA_BLOCKS 65535
#define MAX_CUDA_THREADS 1024
#define PI 3.141592653

__global__ void cuda_calc_pi_step1(int n, int *circle, float *x, float *y)
{
    extern __shared__ int sdata[];
    float c;
    int t = 0;

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    c = (x[i] * x[i]) + (y[i] * y[i]);
    if (c < 0.25f && i<n) t = 1;
    sdata[tid] = t;
    __syncthreads();
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
    if (tid < s) {
    sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0){
        circle[blockIdx.x] = sdata[0];
    };
}

__global__ void cuda_calc_pi_step2(int n, int *circle){
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    sdata[tid] = circle[tid];

    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s = 1; s < blockDim.x; s *= 2){
        if (tid % (2 * s) == 0){
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0){
        circle[blockIdx.x] = sdata[0];
    };
}

/* Function : generate_random_numbers
 * Generates n random numbers for both x and y on the host
 */
void generate_random_numbers(int n, float *x, float *y)
{
    srand(time(NULL));
    for (int i = 0; i < n; i++)
    {
        x[i] = ((float)rand() / RAND_MAX) - 0.5f;
        y[i] = ((float)rand() / RAND_MAX) - 0.5f;
    }
}



/* Function : calculate_pi_monte_carlo
 * Calculates pi on the host by using the monte carlo method, by using a set of
 * random points within a 2R square about point (0,0) we can calculate pi by
 * calculating the ratio of points within the a circle with radius R starting
 * from point (0,0) compared to that of the square. This is done on the host.
 */
float calculate_pi_monte_carlo(int n, float *x, float *y)
{
    int circle = 0;
    for (int i = 0; i < n; i++)
    {
        if (pow(x[i], 2) + pow(y[i], 2) < pow(0.5f, 2))
        {
            circle++;
        }
    }
    return (4.0f * circle) / n;
}

double gpu_calc_pi_monte_carlo(int samples){
    int threads, blocks;

    //You can only do so many samples in an execution due to limitations of the card
    if(samples > (MAX_CUDA_THREADS*MAX_CUDA_THREADS)){
        printf("Too many samples\n");
        return 0.0f;
    }
    threads = (samples < MAX_CUDA_THREADS) ? samples : MAX_CUDA_THREADS;
    blocks = ((samples-1)/MAX_CUDA_THREADS)+1;

    int *d_circle_count, *circle_count;
    float *x, *y, *d_x, *d_y;
    double pi;

    // Allocate memory for our random numbers
    x = (float *)malloc(samples * sizeof(float));
    y = (float *)malloc(samples * sizeof(float));
    circle_count = (int *)malloc(blocks * sizeof(int));

    // Allocate memory on our GPU
    hipMalloc(&d_x, samples * sizeof(float));
    hipMalloc(&d_y, samples * sizeof(float));
    hipMalloc(&d_circle_count,blocks * sizeof(int));

    generate_random_numbers(samples, x, y);

    hipMemcpy(d_x, x, samples * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, samples * sizeof(float), hipMemcpyHostToDevice);

    cuda_calc_pi_step1<<<blocks,threads,threads*sizeof(int)>>>(samples, d_circle_count, d_x, d_y);
    cuda_calc_pi_step2<<<1,blocks,blocks*sizeof(int)>>>(blocks, d_circle_count);

    hipMemcpy(circle_count, d_circle_count, blocks * sizeof(int), hipMemcpyDeviceToHost);

    pi = 4.0 * (double)circle_count[0] / (double)samples;

    // Free memory
    free(x);
    free(y);
    free(circle_count);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_circle_count);

    return pi;
}

int main(void)
{
    // Initiate variables
    double pi;
    int it;
    long int N = 1048576;
    pi=0.0;

    for(int i=0; i<10024;i++){
        if(i==0)hipProfilerStart();
        pi += gpu_calc_pi_monte_carlo(N);
        it = i+1;
        if (i%50==1){
            printf("Samples/1 mill : %d, ",it);
            printf("Pi Estimated : %f, ", pi/it);
            printf("Error : %f\n", (PI-(pi/it))/PI);
        } 
        if(i==100)hipProfilerStop();
    }
    pi /= 10024;
    printf("Pi is estimated to be %f\n", pi);
}