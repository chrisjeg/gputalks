
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define PI 3.141592653
/* Function : generate_random_numbers
 * Generates n random numbers for both x and y on the host
 */
void generate_random_numbers(int n, float*x, float*y){
	srand(time(NULL));
	for (int i = 0; i < n; i++) {
		x[i] = ((float)rand() / RAND_MAX) - 0.5f;
		y[i] = ((float)rand() / RAND_MAX) - 0.5f;
	}
}

/* Function : calculate_pi_monte_carlo
 * Calculates pi on the host by using the monte carlo method, by using a set of
 * random points within a 2R square about point (0,0) we can calculate pi by
 * calculating the ratio of points within the a circle with radius R starting
 * from point (0,0) compared to that of the square. This is done on the host.
 */
float calculate_pi_monte_carlo(int n, float*x, float*y){
	int circle = 0;
	for(int i = 0; i < n; i++) {
		if( pow(x[i],2) + pow(y[i],2)  < pow(0.5f,2) ){
			circle++;
		}
	}
	return (4.0f*circle)/n;
}

int main(void)
{
    // Initiate variables
    int N = 1048576;
    int it;
    float *x, *y;
    double pi;
    pi=0.0;

    // Allocate memory for our random numbers
    x = (float*)malloc(N*sizeof(float));
    y = (float*)malloc(N*sizeof(float));


    for(int i=0; i<10024;i++){
        generate_random_numbers(N, x, y);
        pi += calculate_pi_monte_carlo(N,x,y);
        it = i+1;
        if (i%50==1){
            printf("Samples : %d, ", (N*it));
            printf("Pi Estimated : %f, ", pi/it);
            printf("Error : %f\n", (PI-(pi/it))/PI);
        } 
    }
    pi /= 1024;
    printf("Pi is estimated to be %f\n", pi);

    // Convert timing to a readable format (ms)
    //random_time = rt * 1000 / CLOCKS_PER_SEC;
    //monte_carlo_time = mct * 1000 / CLOCKS_PER_SEC;
    // Free memory
    free(x);
    free(y);
}