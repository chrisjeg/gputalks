
#include <hip/hip_runtime.h>
#include <stdio.h>

void cpu_saxpy(int n, float a, float*x, float *y)
{
  for(int i = 0; i < n; i++){
    y[i] = a*x[i] + y[i];
  }
}

int main(void)
{
  int N = 1<<20;
  float *x, *y;

  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));




  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

 


  // Perform SAXPY on 1M elements
  cpu_saxpy(N, 2.0f, x, y);



  free(x);
  free(y);


}